/*
to compile:
nvcc --gpu-architecture=compute_70 gpu_report_time.cu

requirement for reduction in the device
Hardware: kepler or newer architecture
nvcc: cuda 9 or newer
*/

// Comment out if you excute reduction in the host
#define ACCUMULATE_IN_DEVICE


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

const int N = (256 * 256);
const int BSIZE = 32;

__global__ void gpu_kernel(float *d_A, float *d_B, float *d_C, int len)
{
    // write your code here
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float mysum = 0.0;
    if (i < len) {
        d_C[i] = 0.0;
        for (int j = 0; j < len; j++) {
            mysum += (d_A[i] - d_B[j]) * (d_A[i] - d_B[j]); 
        }
    }
    __syncthreads();
    #ifdef ACCUMULATE_IN_DEVICE
    for (int offset = 32/2; offset > 0; offset >>= 1){
        mysum += __shfl_down_sync(0xffffffff, mysum, offset, 32);
    }
    if (threadIdx.x == 0) d_C[blockIdx.x] = mysum;
    #else
    d_C[i] = mysum;
    #endif
}

__host__ void cpu_kernel(float *d_A, float *d_B, float *d_C, int len)
{
    for (int i = 0; i < len; i++) {
        d_C[i] = 0.0;
        for (int j = 0; j < len; j++) {
            d_C[i] += (d_A[i] - d_B[j]) * (d_A[i] - d_B[j]); 
        }
    }
}

int main(int argc, char **argv)
{
    float *h_A, *h_B, *h_C;  // for host memory
    float *d_A, *d_B, *d_C;  // for device memory
    float result;     // resut
    dim3 grid(N/BSIZE, 1, 1), block(BSIZE, 1, 1); // grid and block size
    hipEvent_t start, stop;              // for measument time on GPU
    struct timeval start_time, end_time;  // for measument time on CPU
    float elapsed_time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* host memory allocation */
    h_A = (float *)malloc(sizeof(float) * N);
    h_B = (float *)malloc(sizeof(float) * N);
    h_C = (float *)malloc(sizeof(float) * N);

    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0f; h_B[i] = 2.0f; h_C[i] = 0.0f;
    }

    /* device memory allocation */
    hipMalloc((void **)&d_A, sizeof(float) * N);
    hipMalloc((void **)&d_B, sizeof(float) * N);
    hipMalloc((void **)&d_C, sizeof(float) * N);

    /* copy data the host to the device */
    hipMemcpy(d_A, h_A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * N, hipMemcpyHostToDevice);

    /* The host calles the karnel */ 
    hipEventRecord(start, 0);

    gpu_kernel<<<grid, block>>>(d_A, d_B, d_C, N);
    hipMemcpy(h_C, d_C, sizeof(float) * N, hipMemcpyDeviceToHost);
    result = 0.0;
    #ifdef ACCUMULATE_IN_DEVICE
    for (int i = 0; i < N/BSIZE; ++i) {
        result += h_C[i];
    }
    #else
    for (int i = 0; i < N; ++i) {
        result += h_C[i];
    }
    #endif
    result /= (float) N;
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    /*  Result write back */

    /*  Release device memory */
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    /* check the result for GPU */
    printf("GPU: result = %f, time = %f [msec]\n", result, elapsed_time);

    /* check the result for CPU */
    gettimeofday(&start_time, NULL);

    cpu_kernel(h_A, h_B, h_C, N);
    result = 0.0;
    for (int i = 0; i < N; ++i)
        result += h_C[i];
    result /= (float) N;

    gettimeofday(&end_time, NULL);
    elapsed_time = (end_time.tv_sec - start_time.tv_sec) * 1000.0 +
                   (end_time.tv_usec - start_time.tv_usec) / 1000.0;
    printf("CPU: result = %f, time = %f [msec]\n", result, elapsed_time);

    /*  Release host memory  */
    free(h_A); free(h_B); free(h_C);

  return 0;
}
